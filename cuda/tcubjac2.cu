#include "hip/hip_runtime.h"
#include "timer.h"
#include "wnrme.h"

#include <hip/hip_runtime.h>
#include "device_code.h"

int main(int argc, char *argv[])
{
  if (6 != argc) {
    (void)fprintf(stderr, "%s device {C|c|D|d|S|s|Z|z} filename lg(batch_size) #batches\n", *argv);
    return EXIT_FAILURE;
  }

  const int dev = atoi(argv[1u]);
  int aux = 0;
  if (hipSuccess != hipGetDeviceCount(&aux))
    return EXIT_FAILURE;
  (void)fprintf(stderr, "CUDA device %d/%d ", dev, aux);
  hipDeviceProp_t prop;
  if ((dev < 0) || (dev >= aux) || (hipSuccess != hipGetDeviceProperties(&prop, dev)) || (hipSuccess != hipGetDevice(&aux)) || ((dev != aux) && (hipSuccess != hipSetDevice(dev)))) {
    (void)fprintf(stderr, "not available!\n");
    aux = EXIT_FAILURE;
  }
  else {
    (void)fprintf(stderr, "[%s] with the warp size %d, ", prop.name, prop.warpSize);
    aux = 0;
  }
  (void)fflush(stderr);
  if (aux)
    return aux;
  const unsigned tpb = (unsigned)(prop.warpSize << 1);

  int kind = toupper(*(argv[2u]));
  (void)fprintf(stderr, "%c ", (char)kind);
  switch ((char)kind)
  {
  case 'C':
    kind = (int)sizeof(float);
    break;
  case 'D':
    kind = -(int)sizeof(double);
    break;
  case 'S':
    kind = -(int)sizeof(float);
    break;
  case 'Z':
    kind = (int)sizeof(double);
    break;
  default:
    (void)fprintf(stderr, "the first argument is invalid.\n");
    return EXIT_FAILURE;
  }

  const size_t n = ((size_t)1u << atoz(argv[4u]));
  (void)fprintf(stderr, "%llu.\n", n);
  (void)fflush(stderr);

  if (n % tpb) {
    (void)fprintf(stderr, "batch size has to be a multiple of %u.\n", tpb);
    return EXIT_FAILURE;
  }
  if ((n / tpb) > (1u << 30u)) {
    (void)fprintf(stderr, "(batch size) / (warp size) has to be at most 2^31.\n");
    return EXIT_FAILURE;
  }
  const unsigned bpg = (unsigned)(n / tpb);

  const size_t b = atoz(argv[5u]);
  if (!b)
    return EXIT_SUCCESS;

  const size_t
    nl = strlen(argv[3u]),
    nl1 = (nl + 1u);
#ifdef _LARGEFILE64_SOURCE
  const int fm = O_RDONLY | O_LARGEFILE;
  char fn[nl + 3u];
#else /* !_LARGEFILE64_SOURCE */
  const int fm = _O_RDONLY | _O_BINARY;
  char *const fn = (char*)_alloca(nl + 3u);
#endif /* ?_LARGEFILE64_SOURCE */
  (void)strcpy((strcpy(fn, argv[3u]) + nl), ". ");

  fn[nl1] = 'k';
  const int fk = open(fn, fm);
  if (-1 >= fk) {
    (void)fprintf(stderr, "Cannot open %s for reading!\n", fn);
    return EXIT_FAILURE;
  }
  fn[nl1] = 'l';
  const int fl = open(fn, fm);
  if (-1 >= fl) {
    (void)fprintf(stderr, "Cannot open %s for reading!\n", fn);
    return EXIT_FAILURE;
  }
  fn[nl1] = 'f';
  const int ff = open(fn, fm);
  if (-1 >= ff) {
    (void)fprintf(stderr, "Cannot open %s for reading!\n", fn);
    return EXIT_FAILURE;
  }
  fn[nl1] = 'g';
  const int fg = open(fn, fm);
  if (-1 >= fg) {
    (void)fprintf(stderr, "Cannot open %s for reading!\n", fn);
    return EXIT_FAILURE;
  }
  fn[nl1] = 'h';
  const int fh = ((kind > 0) ? 0 : open(fn, fm));
  if (-1 >= fh) {
    (void)fprintf(stderr, "Cannot open %s for reading!\n", fn);
    return EXIT_FAILURE;
  }
  fn[nl1] = 'r';
  const int fr = ((kind > 0) ? open(fn, fm) : 0);
  if (-1 >= fr) {
    (void)fprintf(stderr, "Cannot open %s for reading!\n", fn);
    return EXIT_FAILURE;
  }
  fn[nl1] = 'j';
  const int fj = ((kind > 0) ? open(fn, fm) : 0);
  if (-1 >= fj) {
    (void)fprintf(stderr, "Cannot open %s for reading!\n", fn);
    return EXIT_FAILURE;
  }

  wide *const w = (wide*)malloc(n * sizeof(wide));
  if (!w)
    return EXIT_FAILURE;
  const size_t cnt = n * abs(kind);

  void
    *ha11 = NULL, *ha22 = NULL, *ha21r = NULL, *ha21i = NULL,
    *hc = NULL, *hcat = NULL, *hsat = NULL, *hl1 = NULL, *hl2 = NULL;

  if (hipSuccess != hipHostMalloc(&ha11, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostMalloc(&ha22, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostMalloc(&ha21r, cnt))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipHostMalloc(&ha21i, cnt)))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostMalloc(&hc, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostMalloc(&hcat, cnt))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipHostMalloc(&hsat, cnt)))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostMalloc(&hl1, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostMalloc(&hl2, cnt))
    return EXIT_FAILURE;

  void
    *da11 = NULL, *da22 = NULL, *da21r = NULL, *da21i = NULL,
    *dc = NULL, *dcat = NULL, *dsat = NULL, *dl1 = NULL, *dl2 = NULL;

  if (hipSuccess != hipMalloc(&da11, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipMalloc(&da22, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipMalloc(&da21r, cnt))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipMalloc(&da21i, cnt)))
    return EXIT_FAILURE;
  if (hipSuccess != hipMalloc(&dc, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipMalloc(&dcat, cnt))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipMalloc(&dsat, cnt)))
    return EXIT_FAILURE;
  if (hipSuccess != hipMalloc(&dl1, cnt))
    return EXIT_FAILURE;
  if (hipSuccess != hipMalloc(&dl2, cnt))
    return EXIT_FAILURE;

  uint64_t be[2u] = { UINT64_C(0), UINT64_C(0) };
  const uint64_t hz = tsc_get_freq_hz_(&aux);

  (void)fprintf(stdout, "\"B\",\"Ts\",\"ORT\",\"REN\",\"RLN\",\"RLX\",\"RLM\"\n");
  (void)fflush(stdout);
  const char *bf = (const char*)NULL;
  if (b <= 10u)
    bf = "%1llu";
  else if (b <= 100u)
    bf = "%2llu";
  else if (b <= 1000u)
    bf = "%3llu";
  else // b > 1000
    bf = "%llu";
  char a[26u] = { '\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0','\0' };

  for (size_t j = 0u; j < b; ++j) {
    (void)fprintf(stdout, bf, j);
    (void)fflush(stdout);
    if (cnt != (size_t)read(ff, ha11, cnt))
      return EXIT_FAILURE;
    if (hipSuccess != hipMemcpyAsync(da11, ha11, cnt, hipMemcpyHostToDevice))
      return EXIT_FAILURE;
    if (cnt != (size_t)read(fg, ha22, cnt))
      return EXIT_FAILURE;
    if (hipSuccess != hipMemcpyAsync(da22, ha22, cnt, hipMemcpyHostToDevice))
      return EXIT_FAILURE;
    if (kind > 0) {
      if (cnt != (size_t)read(fr, ha21r, cnt))
        return EXIT_FAILURE;
      if (hipSuccess != hipMemcpyAsync(da21r, ha21r, cnt, hipMemcpyHostToDevice))
        return EXIT_FAILURE;
      if (cnt != (size_t)read(fj, ha21i, cnt))
        return EXIT_FAILURE;
      if (hipSuccess != hipMemcpyAsync(da21i, ha21i, cnt, hipMemcpyHostToDevice))
        return EXIT_FAILURE;
    }
    else {
      if (cnt != (size_t)read(fh, ha21r, cnt))
        return EXIT_FAILURE;
      if (hipSuccess != hipMemcpyAsync(da21r, ha21r, cnt, hipMemcpyHostToDevice))
        return EXIT_FAILURE;
    }
    if (hipSuccess != hipDeviceSynchronize())
      return EXIT_FAILURE;
    (void)fprintf(stdout, ",");
    (void)fflush(stdout);
    be[0u] = rdtsc_beg(&aux);
    if (kind > 0) {
      if (kind == (int)sizeof(float)) {
        ccjac2<<<bpg,tpb>>>((const float*)da11, (const float*)da22, (const float*)da21r, (const float*)da21i, (float*)dc, (float*)dcat, (float*)dsat, (float*)dl1, (float*)dl2);
      }
      else {
        zcjac2<<<bpg,tpb>>>((const double*)da11, (const double*)da22, (const double*)da21r, (const double*)da21i, (double*)dc, (double*)dcat, (double*)dsat, (double*)dl1, (double*)dl2);
      }
    }
    else if (kind == -(int)sizeof(float)) {
      scjac2<<<bpg,tpb>>>((const float*)da11, (const float*)da22, (const float*)da21r, (float*)dc, (float*)dcat, (float*)dl1, (float*)dl2);
    }
    else {
      dcjac2<<<bpg,tpb>>>((const double*)da11, (const double*)da22, (const double*)da21r, (double*)dc, (double*)dcat, (double*)dl1, (double*)dl2);
    }
    if (hipSuccess != hipDeviceSynchronize())
      return EXIT_FAILURE;
    be[1u] = rdtsc_end(&aux);
    (void)fprintf(stdout, "%15.9Lf", tsc_lap(hz, be[0u], be[1u]));
    (void)fflush(stdout);
    if (hipSuccess != hipMemcpyAsync(hc, dc, cnt, hipMemcpyDeviceToHost))
      return EXIT_FAILURE;
    if (hipSuccess != hipMemcpyAsync(hcat, dcat, cnt, hipMemcpyDeviceToHost))
      return EXIT_FAILURE;
    if ((kind > 0) && (hipSuccess != hipMemcpyAsync(hsat, dsat, cnt, hipMemcpyDeviceToHost)))
      return EXIT_FAILURE;
    if (hipSuccess != hipMemcpyAsync(hl1, dl1, cnt, hipMemcpyDeviceToHost))
      return EXIT_FAILURE;
    if (hipSuccess != hipMemcpyAsync(hl2, dl2, cnt, hipMemcpyDeviceToHost))
      return EXIT_FAILURE;
    if (hipSuccess != hipDeviceSynchronize())
      return EXIT_FAILURE;
    (void)fprintf(stdout, ",");
    (void)fflush(stdout);
    wide o = W_ZERO, r = W_ZERO;
    if (kind > 0) {
      if (kind == (int)sizeof(float)) {
#ifdef _OPENMP
#pragma omp parallel for default(none) shared(n,ha11,ha22,ha21r,ha21i,hc,hcat,hsat,hl1,hl2,w) reduction(max:o,r)
#endif /* _OPENMP */
        for (size_t i = 0u; i < n; ++i) {
          const wide CS = ((const float*)hc)[i];
          const wide SNR = ((const float*)hcat)[i];
          const wide SNI = ((const float*)hsat)[i];
          wide AE = W_ZERO, AN = W_ZERO;
          o = fmaxw(o, (w[i] = worc(CS, SNR, SNI)));
          r = fmaxw(r, wrec(((const float*)ha11)[i], ((const float*)ha22)[i], ((const float*)ha21r)[i], ((const float*)ha21i)[i], CS, SNR, SNI, ((const float*)hl1)[i], ((const float*)hl2)[i], &AE, &AN));
        }
      }
      else {
#ifdef _OPENMP
#pragma omp parallel for default(none) shared(n,ha11,ha22,ha21r,ha21i,hc,hcat,hsat,hl1,hl2,w) reduction(max:o,r)
#endif /* _OPENMP */
        for (size_t i = 0u; i < n; ++i) {
          const wide CS = ((const double*)hc)[i];
          const wide SNR = ((const double*)hcat)[i];
          const wide SNI = ((const double*)hsat)[i];
          wide AE = W_ZERO, AN = W_ZERO;
          o = fmaxw(o, (w[i] = worc(CS, SNR, SNI)));
          r = fmaxw(r, wrec(((const double*)ha11)[i], ((const double*)ha22)[i], ((const double*)ha21r)[i], ((const double*)ha21i)[i], CS, SNR, SNI, ((const double*)hl1)[i], ((const double*)hl2)[i], &AE, &AN));
        }
      }
    }
    else if (kind == -(int)sizeof(float)) {
#ifdef _OPENMP
#pragma omp parallel for default(none) shared(n,ha11,ha22,ha21r,hc,hcat,hl1,hl2) reduction(max:o,r)
#endif /* _OPENMP */
      for (size_t i = 0u; i < n; ++i) {
        const wide CS = ((const float*)hc)[i];
        const wide SN = ((const float*)hcat)[i];
        wide AE = W_ZERO, AN = W_ZERO;
        o = fmaxw(o, worr(CS, SN));
        r = fmaxw(r, wrer(((const float*)ha11)[i], ((const float*)ha22)[i], ((const float*)ha21r)[i], CS, SN, ((const float*)hl1)[i], ((const float*)hl2)[i], &AE, &AN));
      }
    }
    else {
#ifdef _OPENMP
#pragma omp parallel for default(none) shared(n,ha11,ha22,ha21r,hc,hcat,hl1,hl2) reduction(max:o,r)
#endif /* _OPENMP */
      for (size_t i = 0u; i < n; ++i) {
        const wide CS = ((const double*)hc)[i];
        const wide SN = ((const double*)hcat)[i];
        wide AE = W_ZERO, AN = W_ZERO;
        o = fmaxw(o, worr(CS, SN));
        r = fmaxw(r, wrer(((const double*)ha11)[i], ((const double*)ha22)[i], ((const double*)ha21r)[i], CS, SN, ((const double*)hl1)[i], ((const double*)hl2)[i], &AE, &AN));
      }
    }
    (void)fprintf(stdout, "%s,", dtoa(a, (double)o));
    (void)fprintf(stdout, "%s", dtoa(a, (double)r));
    (void)fflush(stdout);
    size_t ix = n;
    if (kind > 0) {
#ifdef _OPENMP
#pragma omp parallel for default(none) shared(n,o,w) reduction(min:ix)
#endif /* _OPENMP */
      for (size_t i = 0u; i < n; ++i)
        if (w[i] == o)
          ix = i;
      (void)fprintf(stderr, bf, j);
      (void)fprintf(stderr, ",%11llu,%s;", ix, dtoa(a, (double)o));
      if (kind == (int)sizeof(float)) {
        (void)fprintf(stderr, "%s,", dtoa(a, ((const float*)ha11)[ix]));
        (void)fprintf(stderr, "%s,", dtoa(a, ((const float*)ha22)[ix]));
        (void)fprintf(stderr, "(%s,", dtoa(a, ((const float*)ha21r)[ix]));
        (void)fprintf(stderr, "%s);", dtoa(a, ((const float*)ha21i)[ix]));
        (void)fprintf(stderr, "%s,", dtoa(a, ((const float*)hc)[ix]));
        (void)fprintf(stderr, "(%s,", dtoa(a, ((const float*)hcat)[ix]));
        (void)fprintf(stderr, "%s);", dtoa(a, ((const float*)hsat)[ix]));
        (void)fprintf(stderr, "%s,", dtoa(a, ((const float*)hl1)[ix]));
        (void)fprintf(stderr, "%s\n", dtoa(a, ((const float*)hl2)[ix]));
      }
      else {
        (void)fprintf(stderr, "%s,", dtoa(a, ((const double*)ha11)[ix]));
        (void)fprintf(stderr, "%s,", dtoa(a, ((const double*)ha22)[ix]));
        (void)fprintf(stderr, "(%s,", dtoa(a, ((const double*)ha21r)[ix]));
        (void)fprintf(stderr, "%s);", dtoa(a, ((const double*)ha21i)[ix]));
        (void)fprintf(stderr, "%s,", dtoa(a, ((const double*)hc)[ix]));
        (void)fprintf(stderr, "(%s,", dtoa(a, ((const double*)hcat)[ix]));
        (void)fprintf(stderr, "%s);", dtoa(a, ((const double*)hsat)[ix]));
        (void)fprintf(stderr, "%s,", dtoa(a, ((const double*)hl1)[ix]));
        (void)fprintf(stderr, "%s\n", dtoa(a, ((const double*)hl2)[ix]));
      }
      (void)fflush(stderr);
    }
    if (cnt != (size_t)read(fk, hc, cnt))
      return EXIT_FAILURE;
    if (cnt != (size_t)read(fl, hcat, cnt))
      return EXIT_FAILURE;
    (void)fprintf(stdout, ",");
    (void)fflush(stdout);
    wide x = W_ZERO, m = W_ZERO;
    r = W_ZERO;
    if (abs(kind) == (int)sizeof(float)) {
#ifdef _OPENMP
#pragma omp parallel for default(none) shared(n,hl1,hl2,hc,hcat) reduction(max:r,x,m)
#endif /* _OPENMP */
      for (size_t i = 0u; i < n; ++i) {
        wide AE = W_ZERO, AN = W_ZERO;
        const wide RE = wlam(((const float*)hl1)[i], ((const float*)hl2)[i], ((const float*)hc)[i], ((const float*)hcat)[i], &AE, &AN);
        r = fmaxw(r, RE);
        x = fmaxw(x, AE);
        m = fmaxw(m, AN);
      }
    }
    else {
#ifdef _OPENMP
#pragma omp parallel for default(none) shared(n,hl1,hl2,hc,hcat) reduction(max:r,x,m)
#endif /* _OPENMP */
      for (size_t i = 0u; i < n; ++i) {
        wide AE = W_ZERO, AN = W_ZERO;
        const wide RE = wlam(((const double*)hl1)[i], ((const double*)hl2)[i], ((const double*)hc)[i], ((const double*)hcat)[i], &AE, &AN);
        r = fmaxw(r, RE);
        x = fmaxw(x, AE);
        m = fmaxw(m, AN);
      }
    }
    (void)fprintf(stdout, "%s,", dtoa(a, (double)r));
    (void)fprintf(stdout, "%s,", dtoa(a, (double)x));
    (void)fprintf(stdout, "%s\n", dtoa(a, (double)m));
    (void)fflush(stdout);
  }

  if (hipSuccess != hipFree(dl2))
    return EXIT_FAILURE;
  if (hipSuccess != hipFree(dl1))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipFree(dsat)))
    return EXIT_FAILURE;
  if (hipSuccess != hipFree(dcat))
    return EXIT_FAILURE;
  if (hipSuccess != hipFree(dc))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipFree(da21i)))
    return EXIT_FAILURE;
  if (hipSuccess != hipFree(da21r))
    return EXIT_FAILURE;
  if (hipSuccess != hipFree(da22))
    return EXIT_FAILURE;
  if (hipSuccess != hipFree(da11))
    return EXIT_FAILURE;

  if (hipSuccess != hipHostFree(hl2))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostFree(hl1))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipHostFree(hsat)))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostFree(hcat))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostFree(hc))
    return EXIT_FAILURE;
  if ((kind > 0) && (hipSuccess != hipHostFree(ha21i)))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostFree(ha21r))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostFree(ha22))
    return EXIT_FAILURE;
  if (hipSuccess != hipHostFree(ha11))
    return EXIT_FAILURE;

  free(w);

  if (kind > 0) {
    (void)close(fj);
    (void)close(fr);
  }
  else /* real */
    (void)close(fh);
  (void)close(fg);
  (void)close(ff);
  (void)close(fl);
  (void)close(fk);

  return EXIT_SUCCESS;
}
